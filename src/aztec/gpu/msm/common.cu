#include "kernel.cu"
#include <iostream>
#include <vector>

namespace pippenger_common {

/**
 * Execute bucket method
 */ 
template <class point_t, class scalar_t>
point_t* pippenger_t<point_t, scalar_t>::execute_bucket_method(
pippenger_t &config, scalar_t *scalars, point_t *points, unsigned bitsize, unsigned c, size_t npoints, hipStream_t stream) {
    // Initialize dynamic cub_routines object
    config.params = new cub_routines();

    // Bucket initialization kernel
    point_t *buckets;
    unsigned NUM_THREADS = 1 << 10; 

    unsigned NUM_BLOCKS = (config.num_buckets + NUM_THREADS - 1) / NUM_THREADS;
    CUDA_WRAPPER(hipMallocAsync(&buckets, config.num_buckets * 3 * 4 * sizeof(uint64_t), stream));
    initialize_buckets_kernel<<<NUM_BLOCKS * 4, NUM_THREADS, 0, stream>>>(buckets);    // TODO remove the *4

    // Scalars decomposition kernel
    CUDA_WRAPPER(hipMallocAsync(&(params->bucket_indices), sizeof(unsigned) * npoints * (windows + 1), stream));
    CUDA_WRAPPER(hipMallocAsync(&(params->point_indices), sizeof(unsigned) * npoints * (windows + 1), stream));
    split_scalars_kernel<<<NUM_POINTS / NUM_THREADS, NUM_THREADS, 0, stream>>>
        (params->bucket_indices + npoints, params->point_indices + npoints, scalars, npoints, windows, c);

    // Execute CUB routines for determining bucket sizes, offsets, etc. 
    execute_cub_routines(config, config.params, stream);

    // Bucket accumulation kernel
    unsigned NUM_THREADS_2 = 1 << 8;
    unsigned NUM_BLOCKS_2 = ((config.num_buckets + NUM_THREADS_2 - 1) / NUM_THREADS_2) * 4; // TODO remove the *4
    accumulate_buckets_kernel<<<NUM_BLOCKS_2, NUM_THREADS_2, 0, stream>>>
        (buckets, params->bucket_offsets, params->bucket_sizes, params->single_bucket_indices, 
        params->point_indices, points, config.num_buckets);

    // Running sum kernel
    point_t *final_sum;
    CUDA_WRAPPER(hipMallocAsync(&final_sum, windows * 3 * 4 * sizeof(uint64_t), stream));
    bucket_running_sum_kernel<<<26, 4, 0, stream>>>(buckets, final_sum, c);

    // Final accumulation kernel
    point_t *res;
    CUDA_WRAPPER(hipMallocManaged(&res, 3 * 4 * sizeof(uint64_t)));
    final_accumulation_kernel<<<1, 4, 0, stream>>>(final_sum, res, windows, c);
    
    // Synchronize stream
    hipStreamSynchronize(stream);

    // Check for errors codes
    auto res1 = hipGetLastError();
    cout << "Cuda Error Code: " << res1 << endl;

    // Free host and device memory 
    CUDA_WRAPPER(hipHostFree(points));
    CUDA_WRAPPER(hipHostFree(scalars));
    CUDA_WRAPPER(hipFreeAsync(buckets, stream));
    CUDA_WRAPPER(hipFreeAsync(params->bucket_indices, stream));
    CUDA_WRAPPER(hipFreeAsync(params->point_indices, stream));
    CUDA_WRAPPER(hipFreeAsync(params->sort_indices_temp_storage, stream));
    CUDA_WRAPPER(hipFreeAsync(params->single_bucket_indices, stream));
    CUDA_WRAPPER(hipFreeAsync(params->bucket_sizes, stream));
    CUDA_WRAPPER(hipFreeAsync(params->nof_buckets_to_compute, stream));
    CUDA_WRAPPER(hipFreeAsync(params->encode_temp_storage, stream));
    CUDA_WRAPPER(hipFreeAsync(params->bucket_offsets, stream));
    CUDA_WRAPPER(hipFreeAsync(params->offsets_temp_storage, stream));
    CUDA_WRAPPER(hipFree(final_sum));
    CUDA_WRAPPER(hipFree(res));

    return res;
}

/**
 * CUB routines referenced from: https://github.com/ingonyama-zk/icicle (inspired by zkSync's era-bellman-cuda library)
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::execute_cub_routines(pippenger_t &config, cub_routines *params, hipStream_t stream) {
    // Radix sort algorithm
    size_t sort_indices_temp_storage_bytes; 
    // TJP - how is the value of npoints found?
    hipcub::DeviceRadixSort::SortPairs(params->sort_indices_temp_storage, sort_indices_temp_storage_bytes, params->bucket_indices 
                                    + npoints, params->bucket_indices, params->point_indices + npoints, params->point_indices, 
                                    npoints, 0, sizeof(unsigned) * 8, stream);
    CUDA_WRAPPER(hipMallocAsync(&(params->sort_indices_temp_storage), sort_indices_temp_storage_bytes, stream));
    for (unsigned i = 0; i < config.windows; i++) {
        unsigned offset_out = i * npoints;
        unsigned offset_in = offset_out + npoints;
        hipcub::DeviceRadixSort::SortPairs(params->sort_indices_temp_storage, sort_indices_temp_storage_bytes, params->bucket_indices 
                                        + offset_in, params->bucket_indices + offset_out, params->point_indices + offset_in, 
                                        params->point_indices + offset_out, npoints, 0, sizeof(unsigned) * 8, stream);
    }

    // Perform length encoding
    CUDA_WRAPPER(hipMallocAsync(&(params->single_bucket_indices), sizeof(unsigned) * config.num_buckets, stream));

    // TODO: THIS ALLOCATION NEEDS TO BE CHANGED AND WILL VARY RUNTIME OF PIPPENGER FOR SOME REASON
    /** Tal: (sizeof(unsigned) * config.num_buckets * config.num_buckets) is definitely not the right memory size, and it should instead 
    be (sizeof(unsigned) * config.num_buckets) or (sizeof(unsigned) * config.num_buckets + 1) .**/
    CUDA_WRAPPER(hipMallocAsync(&(params->bucket_sizes), sizeof(unsigned) * config.num_buckets * config.num_buckets, stream));
    CUDA_WRAPPER(hipMallocAsync(&(params->nof_buckets_to_compute), sizeof(unsigned), stream));
    size_t encode_temp_storage_bytes = 0;
    hipcub::DeviceRunLengthEncode::Encode(params->encode_temp_storage, encode_temp_storage_bytes, params->bucket_indices, 
                                       params->single_bucket_indices, params->bucket_sizes, params->nof_buckets_to_compute, 
                                       config.windows * npoints, stream);
    CUDA_WRAPPER(hipMallocAsync(&(params->encode_temp_storage), encode_temp_storage_bytes, stream));
    hipcub::DeviceRunLengthEncode::Encode(params->encode_temp_storage, encode_temp_storage_bytes, params->bucket_indices, 
                                       params->single_bucket_indices, params->bucket_sizes, params->nof_buckets_to_compute, 
                                       config.windows * npoints, stream);

    // Calculate bucket offsets
    CUDA_WRAPPER(hipMallocAsync(&(params->bucket_offsets), sizeof(unsigned) * config.num_buckets, stream));
    size_t offsets_temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(params->offsets_temp_storage, offsets_temp_storage_bytes, params->bucket_sizes, 
                                  params->bucket_offsets, config.num_buckets, stream);
    CUDA_WRAPPER(hipMallocAsync(&(params->offsets_temp_storage), offsets_temp_storage_bytes, stream));
    hipcub::DeviceScan::ExclusiveSum(params->offsets_temp_storage, offsets_temp_storage_bytes, params->bucket_sizes, 
                                  params->bucket_offsets, config.num_buckets, stream);
}

/**
 * Calculate number of windows and buckets
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::calculate_windows(pippenger_t &config, size_t npoints) {
    config.windows = BITSIZE / C; 
    if (BITSIZE % C) {  
        windows++;
    }
    config.num_buckets = windows << C; 
    config.npoints = npoints;
}

/**
 * Calculate the amount of device storage required to store bases 
 */
template <class point_t, class scalar_t>
size_t pippenger_t<point_t, scalar_t>::get_size_bases(pippenger_t &config) {
    return NUM_POINTS * sizeof(point_t);
}

/**
 * Calculate the amount of device storage required to store scalars 
 */
template <class point_t, class scalar_t>
size_t pippenger_t<point_t, scalar_t>::get_size_scalars(pippenger_t &config) {
    // return config.n * sizeof(scalar_t);
    return NUM_POINTS * sizeof(scalar_t);
}

/**
 * Allocate device storage for bases
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::allocate_bases(pippenger_t &config) {
    device_base_ptrs.allocate(get_size_bases(config));
}

/**
 * Allocate device storage for scalars
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::allocate_scalars(pippenger_t &config) {
    device_scalar_ptrs.allocate(get_size_scalars(config));
}

/**
 * Transfer base points to GPU device
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::transfer_bases_to_device(
pippenger_t &config, point_t *device_bases_ptrs, const point_t *points, hipStream_t stream) {    
    CUDA_WRAPPER(hipMemcpyAsync(device_bases_ptrs, points, NUM_POINTS * LIMBS * sizeof(uint64_t), hipMemcpyHostToDevice, stream));
}

/**
 * Transfer scalars to GPU device
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::transfer_scalars_to_device(
pippenger_t &config, scalar_t *device_scalar_ptrs, fr *scalars, hipStream_t stream) {
    CUDA_WRAPPER(hipMemcpyAsync(device_scalar_ptrs, scalars, NUM_POINTS * LIMBS * sizeof(uint64_t), hipMemcpyHostToDevice, stream));
}

/**
 * Allocate pinned memory using hipHostMalloc
 */
template <class T>
void device_ptr<T>::allocate(size_t bytes) {
    T* d_ptr;
    CUDA_WRAPPER(hipHostMalloc(&d_ptr, bytes));
    d_ptrs.push_back(d_ptr);
}

/**
 * Get size of d_ptrs vector
 */
template <class T>
size_t device_ptr<T>::size() {
    return d_ptrs.size();
}

/**
 * Operator overloading for device_ptr indexing
 */
template <class T>
T* device_ptr<T>::operator[](size_t i) {
    if (i > d_ptrs.size() - 1) {
        cout << "Indexing error!" << endl;
        throw;
    }
    return d_ptrs[i];
}

/**
 * Verify results
 */ 
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::verify_result(point_t *result_1, point_t **result_2) {
    var *result;
    CUDA_WRAPPER(hipMallocManaged(&result, LIMBS * sizeof(uint64_t)));
    comparator_kernel<<<1, 4>>>(result_1, result_2[0], result);
    hipDeviceSynchronize();

    assert (result[0] == 1);
    assert (result[1] == 1);
    assert (result[2] == 1);
    assert (result[3] == 1);

    cout << "MSM Result Verified!" << endl;
}

/**
 * Print results
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::print_result(g1_gpu::element *result_1, g1_gpu::element **result_2) {
    for (int i = 0; i < LIMBS; i++) {
        printf("result_naive_msm is: %zu\n", result_1[0].x.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_naive_msm is: %zu\n", result_1[0].y.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_naive_msm is: %zu\n", result_1[0].z.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_bucket_method_msm is: %zu\n", result_2[0][0].x.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_bucket_method_msm is: %zu\n", result_2[0][0].y.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_bucket_method_msm is: %zu\n", result_2[0][0].z.data[i]);
    }
}

}