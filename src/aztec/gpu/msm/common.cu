#include "kernel.cu"
#include "reduce_by_key.cu"
#include <chrono>
#include <iostream>
#include <vector>
#include <thrust/execution_policy.h>
#include <thrust/scatter.h>


using namespace std::chrono;

high_resolution_clock::time_point t1, t2;

namespace pippenger_common {

struct padd_functor
{
    __device__ g1_single::element operator()(g1_single::element a, g1_single::element b) const
    {

        g1_single::add(a.x.data,
                        a.y.data,
                        a.z.data,
                        b.x.data,
                        b.y.data,
                        b.z.data,
                        a.x.data,
                        a.y.data,
                        a.z.data);
        return a;
    }
};

/**
 * Execute bucket method
 */ 
template <class point_t, class scalar_t>
point_t* pippenger_t<point_t, scalar_t>::execute_bucket_method(
pippenger_t &config, scalar_t *scalars, point_t *points, unsigned bitsize, unsigned c, size_t npoints, hipStream_t stream) {
    typedef gpu_barretenberg_single::gpu_group_elements_single::element_single<gpu_barretenberg_single::fq_single, gpu_barretenberg_single::fr_single> point_single_t;

    // START TIMING
    hipDeviceSynchronize();
    t1 = high_resolution_clock::now();

    size_t n = npoints * config.windows;
    thrust::device_vector<FlagType> head_flags;
    thrust::device_vector<FlagType> tail_flags;
    thrust::device_vector<FlagType> scanned_tail_flags;
    if (TEST_REDUCE) {
        head_flags = thrust::device_vector<FlagType>(n);
        tail_flags = thrust::device_vector<FlagType>(n);
        scanned_tail_flags = thrust::device_vector<FlagType>(n);
    }
    thrust::device_vector<point_single_t> scanned_values(n);

    // Initialize dynamic cub_routines object
    config.params = new cub_routines();

    point_t *buckets;
    unsigned NUM_THREADS = 1 << 10; 

    unsigned NUM_BLOCKS = (config.num_buckets + NUM_THREADS - 1) / NUM_THREADS;
    CUDA_WRAPPER(hipMallocAsync(&buckets, config.num_buckets * sizeof(point_single_t), stream));
    
    // Bucket initialization kernel
    // initialize_buckets_kernel<<<NUM_BLOCKS * 4, NUM_THREADS, 0, stream>>>(buckets); 
    CUDA_WRAPPER(hipMemsetAsync(buckets, 0, config.num_buckets * sizeof(point_single_t), stream));

    // Scalars decomposition kernel
    CUDA_WRAPPER(hipMallocAsync(&(params->bucket_indices), sizeof(unsigned) * npoints * (windows + 1), stream));
    CUDA_WRAPPER(hipMallocAsync(&(params->point_indices), sizeof(unsigned) * npoints * (windows + 1), stream));
    split_scalars_kernel<<<NUM_POINTS / NUM_THREADS, NUM_THREADS, 0, stream>>>
        (params->bucket_indices + npoints, params->point_indices + npoints, scalars, npoints, windows, c);

    // Execute CUB routines for determining bucket sizes, offsets, etc. 
    execute_cub_routines(config, config.params, stream);

    // // Bucket accumulation kernel
    // unsigned NUM_THREADS_2 = 1 << 8;
    // unsigned NUM_BLOCKS_2 = ((config.num_buckets + NUM_THREADS_2 - 1) / NUM_THREADS_2) * 4;
    // accumulate_buckets_kernel<<<NUM_BLOCKS_2, NUM_THREADS_2, 0, stream>>>
    //     (buckets, params->bucket_offsets, params->bucket_sizes, params->single_bucket_indices, 
    //     params->point_indices, points, config.num_buckets);

    // Bucket accumulation kernel
    point_single_t *buckets_single = reinterpret_cast<point_single_t *>(buckets);
    point_single_t *points_single = reinterpret_cast<point_single_t *>(points);
    auto points_iter = thrust::make_permutation_iterator(points_single, params->point_indices);
    if (TEST_REDUCE) {
        reduce_by_key_into_map(
            thrust::cuda::par_nosync.on(stream),
            params->bucket_indices,
            params->bucket_indices + npoints * config.windows,
            points_iter,
            params->bucket_indices,
            buckets_single,
            thrust::equal_to<>{},
            padd_functor{},
            head_flags,
            tail_flags,
            scanned_tail_flags,
            scanned_values
        );
    } else {
        auto [_, scanned_values_end] = thrust::reduce_by_key(
            thrust::cuda::par_nosync.on(stream),
            params->bucket_indices,
            params->bucket_indices + npoints * config.windows,
            points_iter,
            params->bucket_indices,
            scanned_values.begin(),
            thrust::equal_to<>{},
            padd_functor{}
        );
        thrust::scatter(
            thrust::cuda::par_nosync.on(stream),
            scanned_values.begin(),
            scanned_values_end,
            params->bucket_indices,
            buckets_single
        );
    }

    // Running sum kernel
    point_t *final_sum;
    CUDA_WRAPPER(hipMallocAsync(&final_sum, windows * sizeof(point_single_t), stream));
    bucket_running_sum_kernel<<<config.windows, 4, 0, stream>>>(buckets, final_sum, c);

    // Final accumulation kernel
    point_t *res;
    CUDA_WRAPPER(hipMallocManaged(&res, sizeof(point_single_t)));
    final_accumulation_kernel<<<1, 4, 0, stream>>>(final_sum, res, windows, c);
    
    // Synchronize stream
    hipStreamSynchronize(stream);
    
    // End timer
    t2 = high_resolution_clock::now();

    cout << "Pippenger execution took " << duration_cast<duration<double>>(t2 - t1).count() << endl;

    // Check for errors codes
    auto res1 = hipGetLastError();
    cout << "Cuda Error Code: " << res1 << endl;

    // Free host and device memory 
    CUDA_WRAPPER(hipHostFree(points));
    CUDA_WRAPPER(hipHostFree(scalars));
    CUDA_WRAPPER(hipFreeAsync(buckets, stream));
    CUDA_WRAPPER(hipFreeAsync(params->bucket_indices, stream));
    CUDA_WRAPPER(hipFreeAsync(params->point_indices, stream));
    CUDA_WRAPPER(hipFreeAsync(params->sort_indices_temp_storage, stream));
    // CUDA_WRAPPER(hipFreeAsync(params->single_bucket_indices, stream));
    // CUDA_WRAPPER(hipFreeAsync(params->bucket_sizes, stream));
    // CUDA_WRAPPER(hipFreeAsync(params->nof_buckets_to_compute, stream));
    // CUDA_WRAPPER(hipFreeAsync(params->encode_temp_storage, stream));
    // CUDA_WRAPPER(hipFreeAsync(params->bucket_offsets, stream));
    // CUDA_WRAPPER(hipFreeAsync(params->offsets_temp_storage, stream));
    CUDA_WRAPPER(hipFree(final_sum));
    // CUDA_WRAPPER(hipFree(res));

    return res;
}

/**
 * CUB routines referenced from: https://github.com/ingonyama-zk/icicle (inspired by zkSync's era-bellman-cuda library)
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::execute_cub_routines(pippenger_t &config, cub_routines *params, hipStream_t stream) {
    // Radix sort algorithm
    size_t sort_indices_temp_storage_bytes;

    // sort [npoints:2*npoints] into [0:npoints] by lower 32 bits
    hipcub::DeviceRadixSort::SortPairs(
        params->sort_indices_temp_storage,
        sort_indices_temp_storage_bytes,
        params->bucket_indices + npoints, // keys_in
        params->bucket_indices,           // keys_out
        params->point_indices + npoints,  // values_in
        params->point_indices,            // values_out
        npoints,                          // num_items
        0,                                // begin_bit
        sizeof(unsigned) * 8,             // end_bit
        stream);
    CUDA_WRAPPER(hipMallocAsync(&(params->sort_indices_temp_storage), sort_indices_temp_storage_bytes, stream));


    for (unsigned i = 0; i < config.windows; i++)
    {
        unsigned offset_out = i * npoints;
        unsigned offset_in = offset_out + npoints;

        // sort [(i+1)*npoints:(i+2)*npoints] into [i*npoints:(i+1)*npoints] by lower 32 bits
        hipcub::DeviceRadixSort::SortPairs(
            params->sort_indices_temp_storage,
            sort_indices_temp_storage_bytes,
            params->bucket_indices + offset_in,  // keys_in
            params->bucket_indices + offset_out, // keys_out
            params->point_indices + offset_in,   // values_in
            params->point_indices + offset_out,  // values_out
            npoints,                             // num_items
            0,                                   // start_bit
            sizeof(unsigned) * 8,                // end_bit
            stream);
    }
}

/**
 * Calculate number of windows and buckets
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::calculate_windows(pippenger_t &config, size_t npoints) {
    config.windows = BITSIZE / C; 
    if (BITSIZE % C) {  
        windows++;
    }
    config.num_buckets = windows << C; 
    config.npoints = npoints;
}

/**
 * Calculate the amount of device storage required to store bases 
 */
template <class point_t, class scalar_t>
size_t pippenger_t<point_t, scalar_t>::get_size_bases(pippenger_t &config) {
    return NUM_POINTS * sizeof(point_t);
}

/**
 * Calculate the amount of device storage required to store scalars 
 */
template <class point_t, class scalar_t>
size_t pippenger_t<point_t, scalar_t>::get_size_scalars(pippenger_t &config) {
    // return config.n * sizeof(scalar_t);
    return NUM_POINTS * sizeof(scalar_t);
}

/**
 * Allocate device storage for bases
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::allocate_bases(pippenger_t &config) {
    device_base_ptrs.allocate(get_size_bases(config));
}

/**
 * Allocate device storage for scalars
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::allocate_scalars(pippenger_t &config) {
    device_scalar_ptrs.allocate(get_size_scalars(config));
}

/**
 * Transfer base points to GPU device
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::transfer_bases_to_device(
pippenger_t &config, point_t *device_bases_ptrs, const point_t *points, hipStream_t stream) {    
    CUDA_WRAPPER(hipMemcpyAsync(device_bases_ptrs, points, NUM_POINTS * LIMBS * sizeof(uint64_t), hipMemcpyHostToDevice, stream));
}

/**
 * Transfer scalars to GPU device
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::transfer_scalars_to_device(
pippenger_t &config, scalar_t *device_scalar_ptrs, fr *scalars, hipStream_t stream) {
    CUDA_WRAPPER(hipMemcpyAsync(device_scalar_ptrs, scalars, NUM_POINTS * LIMBS * sizeof(uint64_t), hipMemcpyHostToDevice, stream));
}

/**
 * Allocate pinned memory using hipHostMalloc
 */
template <class T>
void device_ptr<T>::allocate(size_t bytes) {
    T* d_ptr;
    CUDA_WRAPPER(hipHostMalloc(&d_ptr, bytes));
    d_ptrs.push_back(d_ptr);
}

/**
 * Get size of d_ptrs vector
 */
template <class T>
size_t device_ptr<T>::size() {
    return d_ptrs.size();
}

/**
 * Operator overloading for device_ptr indexing
 */
template <class T>
T* device_ptr<T>::operator[](size_t i) {
    if (i > d_ptrs.size() - 1) {
        cout << "Indexing error!" << endl;
        throw;
    }
    return d_ptrs[i];
}

/**
 * Verify results
 */ 
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::verify_result(point_t *result_1, point_t **result_2) {
    var *result;
    CUDA_WRAPPER(hipMallocManaged(&result, LIMBS * sizeof(uint64_t)));
    comparator_kernel<<<1, 4>>>(result_1, result_2[0], result);
    hipDeviceSynchronize();

    assert (result[0] == 1);
    assert (result[1] == 1);
    assert (result[2] == 1);
    assert (result[3] == 1);

    cout << "MSM Result Verified!" << endl;
}

/**
 * Print results
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::print_result(g1_gpu::element *result_1, g1_gpu::element **result_2) {
    for (int i = 0; i < LIMBS; i++) {
        printf("result_naive_msm is: %zu\n", result_1[0].x.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_naive_msm is: %zu\n", result_1[0].y.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_naive_msm is: %zu\n", result_1[0].z.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_bucket_method_msm is: %zu\n", result_2[0][0].x.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_bucket_method_msm is: %zu\n", result_2[0][0].y.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_bucket_method_msm is: %zu\n", result_2[0][0].z.data[i]);
    }
}

}