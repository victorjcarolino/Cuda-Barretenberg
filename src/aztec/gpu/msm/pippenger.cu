#include "pippenger.cuh"
#include <iostream>
#include <memory>
#include <iostream>
#include <fstream>
#include <string>
#include <chrono>

using namespace std;
using namespace std::chrono;

namespace pippenger_common {

/**
 * Entry point into initializing "Pippenger's Bucket" Method
 */ 
template <class P, class S>
Context<point_t, scalar_t> *msm_t<P, S>::pippenger_initialize(g1::affine_element* points, fr *scalars, int num_streams, size_t npoints) {
    try {
        // Initialize 'Context' object 
        Context<point_t, scalar_t> *context = new Context<point_t, scalar_t>();

        // Calculate windows and buckets
        context->pipp.calculate_windows(context->pipp, npoints);

        // Dynamically allocate streams at runtime
        context->pipp.streams = new hipStream_t[num_streams];
        for (int i = 0; i < num_streams; i++) {
            CUDA_WRAPPER(hipStreamCreateWithFlags(&(context->pipp.streams[i]), hipStreamNonBlocking));
        }

        // Allocate GPU storage for elliptic curve bases and scalars 
        for (int i = 0; i < num_streams; i++) { 
            context->pipp.allocate_bases(context->pipp);
            context->pipp.allocate_scalars(context->pipp);
        }

        // Convert affine to jacobian coordinates 
        g1_gpu::affine_element *a_points;
        g1_gpu::element *j_points;
        CUDA_WRAPPER(hipMallocAsync(&j_points, 3 * NUM_POINTS * LIMBS * sizeof(uint64_t), context->pipp.streams[0]));
        CUDA_WRAPPER(hipMallocAsync(&a_points, 2 * NUM_POINTS * LIMBS * sizeof(uint64_t), context->pipp.streams[0]));
        CUDA_WRAPPER(hipMemcpyAsync(a_points, points, NUM_POINTS * LIMBS * 2 * sizeof(uint64_t), 
                                    hipMemcpyHostToDevice, context->pipp.streams[0]));
        affine_to_jacobian<<<(NUM_POINTS / 256), 256, 0, context->pipp.streams[0]>>>(a_points, j_points, NUM_POINTS);
        
        // Transfer bases and scalars to device
        for (int i = 0; i < num_streams; i++) { 
            context->pipp.transfer_scalars_to_device(
                context->pipp, context->pipp.device_scalar_ptrs.d_ptrs[i], scalars, context->pipp.streams[i]
            );
            context->pipp.transfer_bases_to_device(
                context->pipp, context->pipp.device_base_ptrs.d_ptrs[i], j_points, context->pipp.streams[i]
            );
        }

        // Free intermediary state variables
        CUDA_WRAPPER(hipFreeAsync(j_points, context->pipp.streams[0]));
        CUDA_WRAPPER(hipFreeAsync(a_points, context->pipp.streams[0]));
    
        return context;
    }
    catch (hipError_t) {
        cout << "Failed to initialize MSM." << endl;
        throw;
    }
}

/**
 * Perform MSM Double-And-Add Method
 */ 
template <class P, class S>
g1_gpu::element* msm_t<P, S>::msm_double_and_add(
Context<point_t, scalar_t> *context, size_t npoints, g1::affine_element *points, fr *scalars) {
    // Allocate unified memory and launch kernel 
    g1_gpu::element *result;
    CUDA_WRAPPER(hipMallocManaged(&result, 3 * NUM_POINTS * LIMBS * sizeof(uint64_t)));
    double_and_add_kernel<<<1, 4, 0, 0>>>(
        context->pipp.device_scalar_ptrs.d_ptrs[0], context->pipp.device_base_ptrs.d_ptrs[0], result, npoints
    );
    hipDeviceSynchronize();

    // CUDA_WRAPPER(hipFree(result));
    
    return result;
}

/**
 * Perform MSM Bucket Method
 */ 
template <class P, class S>
g1_gpu::element** msm_t<P, S>::msm_bucket_method(
Context<point_t, scalar_t> *context, g1::affine_element *points, fr *scalars, int num_streams) {
    // Start timer
    high_resolution_clock::time_point t1 = high_resolution_clock::now();

    // Launch pippenger kernel
    cout << "starting pippenger!" << endl;
    g1_gpu::element **result = new g1_gpu::element*[num_streams];
    for (int i = 0; i < num_streams; i++) { 
        result[i] = context->pipp.execute_bucket_method(
            context->pipp, context->pipp.device_scalar_ptrs.d_ptrs[i], context->pipp.device_base_ptrs.d_ptrs[i], 
            BITSIZE, C, context->pipp.npoints, context->pipp.streams[i]
        );
    }
    cout << "finished pippenger!" << endl;

    // End timer
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "Pippenger executed in " << time_span.count() << " seconds." << endl;

    return result;
}

}