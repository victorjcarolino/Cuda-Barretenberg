#include "hip/hip_runtime.h"
#include "group.cu"

using namespace std;
using namespace std::chrono;
using namespace gpu_barretenberg;

static constexpr size_t LIMBS_NUM = 4;
static constexpr size_t BLOCKS = 1;
static constexpr size_t THREADS = 1;

/* -------------------------- Mixed Addition Test ---------------------------------------------- */

__global__ void initialize_mixed_add_check_against_constants
(var *a, var *b, var *c, var *x, var *y, var *z, var *expected_x, var *expected_y, var *expected_z) {
    fq_gpu a_x{ 0x92716caa6cac6d26, 0x1e6e234136736544, 0x1bb04588cde00af0, 0x9a2ac922d97e6f5 };
    fq_gpu a_y{ 0x9e693aeb52d79d2d, 0xf0c1895a61e5e975, 0x18cd7f5310ced70f, 0xac67920a22939ad };
    fq_gpu a_z{ 0xfef593c9ce1df132, 0xe0486f801303c27d, 0x9bbd01ab881dc08e, 0x2a589badf38ec0f9 };
    fq_gpu b_x{ 0xa1ec5d1398660db8, 0x6be3e1f6fd5d8ab1, 0x69173397dd272e11, 0x12575bbfe1198886 };
    fq_gpu b_y{ 0xcfbfd4441138823e, 0xb5f817e28a1ef904, 0xefb7c5629dcc1c42, 0x1a9ed3d6f846230e };
    fq_gpu exp_x{ 0x2a9d0201fccca20, 0x36f969b294f31776, 0xee5534422a6f646, 0x911dbc6b02310b6 };
    fq_gpu exp_y{ 0x14c30aaeb4f135ef, 0x9c27c128ea2017a1, 0xf9b7d80c8315eabf, 0x35e628df8add760 };
    fq_gpu exp_z{ 0xa43fe96673d10eb3, 0x88fbe6351753d410, 0x45c21cc9d99cb7d, 0x3018020aa6e9ede5 };

    for (int i = 0; i < LIMBS_NUM; i++) {
        a[i] = a_x.data[i];
        b[i] = a_y.data[i];
        c[i] = a_z.data[i];
        x[i] = b_x.data[i];
        y[i] = b_y.data[i];
        expected_x[i] = exp_x.data[i];
        expected_y[i] = exp_y.data[i];
        expected_z[i] = exp_z.data[i];
    }
}

__global__ void mixed_add_check_against_constants
(var *a, var *b, var *c, var *x, var *y, var *z, var *res_x, var *res_y, var *res_z) {
    g1_gpu::element lhs;
    g1_gpu::affine_element rhs;
    g1_gpu::element result;
    g1_gpu::element expected;
    
    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        lhs.x.data[tid] = fq_gpu::to_monty(a[tid], res_x[tid]);
        lhs.y.data[tid] = fq_gpu::to_monty(b[tid], res_x[tid]);
        lhs.z.data[tid] = fq_gpu::to_monty(c[tid], res_x[tid]);
        rhs.x.data[tid] = fq_gpu::to_monty(x[tid], res_x[tid]);
        rhs.y.data[tid] = fq_gpu::to_monty(y[tid], res_x[tid]);

        // lhs + rhs (affine element + jacobian element)
        g1_gpu::mixed_add(
            lhs.x.data[tid], lhs.y.data[tid], lhs.z.data[tid], 
            rhs.x.data[tid], rhs.y.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // Return results from montgomery form 
        fq_gpu::from_monty(res_x[tid], res_x[tid]);
        fq_gpu::from_monty(res_y[tid], res_y[tid]);
        fq_gpu::from_monty(res_z[tid], res_z[tid]);
    }
}

/* -------------------------- Doubling Test ---------------------------------------------- */

__global__ void initialize_dbl_check_against_constants
(var *a, var *b, var *c, var *x, var *y, var *z, var *expected_x, var *expected_y, var *expected_z) {
    fq_gpu a_x{ 0x8d1703aa518d827f, 0xd19cc40779f54f63, 0xabc11ce30d02728c, 0x10938940de3cbeec };
    fq_gpu a_y{ 0xcf1798994f1258b4, 0x36307a354ad90a25, 0xcd84adb348c63007, 0x6266b85241aff3f };
    fq_gpu a_z{ 0xe213e18fd2df7044, 0xb2f42355982c5bc8, 0xf65cf5150a3a9da1, 0xc43bde08b03aca2 };
    fq_gpu exp_x{ 0xd5c6473044b2e67c, 0x89b185ea20951f3a, 0x4ac597219cf47467, 0x2d00482f63b12c86 };
    fq_gpu exp_y{ 0x4e7e6c06a87e4314, 0x906a877a71735161, 0xaa7b9893cc370d39, 0x62f206bef795a05 };
    fq_gpu exp_z{ 0x8813bdca7b0b115a, 0x929104dffdfabd22, 0x3fff575136879112, 0x18a299c1f683bdca };

    for (int i = 0; i < LIMBS_NUM; i++) {
        a[i] = a_x.data[i];
        b[i] = a_y.data[i];
        c[i] = a_z.data[i];
        expected_x[i] = exp_x.data[i];
        expected_y[i] = exp_y.data[i];
        expected_z[i] = exp_z.data[i];
    }
}

__global__ void dbl_check_against_constants
(var *a, var *b, var *c, var *x, var *y, var *z, var *res_x, var *res_y, var *res_z) {
    g1_gpu::element lhs;
    g1_gpu::element result;
    g1_gpu::element expected;
    
    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        lhs.x.data[tid] = fq_gpu::to_monty(a[tid], res_x[tid]);
        lhs.y.data[tid] = fq_gpu::to_monty(b[tid], res_y[tid]);
        lhs.z.data[tid] = fq_gpu::to_monty(c[tid], res_z[tid]);

        // lhs.doubling
        g1_gpu::doubling(
            lhs.x.data[tid], lhs.y.data[tid], lhs.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );
        //  (lhs.doubling).doubling
        g1_gpu::doubling(
            res_x[tid], res_y[tid], res_z[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );
        //  ((lhs.doubling).doubling).doubling
        g1_gpu::doubling(
            res_x[tid], res_y[tid], res_z[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // Return results from montgomery form 
        fq_gpu::from_monty(res_x[tid], res_x[tid]);
        fq_gpu::from_monty(res_y[tid], res_y[tid]);
        fq_gpu::from_monty(res_z[tid], res_z[tid]);
    }
}

/* -------------------------- Addition Test ---------------------------------------------- */

__global__ void initialize_add_check_against_constants
(var *a, var *b, var *c, var *x, var *y, var *z, var *expected_x, var *expected_y, var *expected_z) {
    fq_gpu a_x{ 0x184b38afc6e2e09a, 0x4965cd1c3687f635, 0x334da8e7539e71c4, 0xf708d16cfe6e14 };
    fq_gpu a_y{ 0x2a6ff6ffc739b3b6, 0x70761d618b513b9, 0xbf1645401de26ba1, 0x114a1616c164b980 };
    fq_gpu a_z{ 0x10143ade26bbd57a, 0x98cf4e1f6c214053, 0x6bfdc534f6b00006, 0x1875e5068ababf2c };
    fq_gpu b_x{ 0xafdb8a15c98bf74c, 0xac54df622a8d991a, 0xc6e5ae1f3dad4ec8, 0x1bd3fb4a59e19b52 };
    fq_gpu b_y{ 0x21b3bb529bec20c0, 0xaabd496406ffb8c1, 0xcd3526c26ac5bdcb, 0x187ada6b8693c184 };
    fq_gpu b_z{ 0xffcd440a228ed652, 0x8a795c8f234145f1, 0xd5279cdbabb05b95, 0xbdf19ba16fc607a };
    fq_gpu exp_x{ 0x18764da36aa4cd81, 0xd15388d1fea9f3d3, 0xeb7c437de4bbd748, 0x2f09b712adf6f18f };
    fq_gpu exp_y{ 0x50c5f3cab191498c, 0xe50aa3ce802ea3b5, 0xd9d6125b82ebeff8, 0x27e91ba0686e54fe };
    fq_gpu exp_z{ 0xe4b81ef75fedf95, 0xf608edef14913c75, 0xfd9e178143224c96, 0xa8ae44990c8accd };

    for (int i = 0; i < LIMBS_NUM; i++) {
        a[i] = a_x.data[i];
        b[i] = a_y.data[i];
        c[i] = a_z.data[i];
        x[i] = b_x.data[i];
        y[i] = b_y.data[i];
        z[i] = b_z.data[i];
        expected_x[i] = exp_x.data[i];
        expected_y[i] = exp_y.data[i];
        expected_z[i] = exp_z.data[i];
    }
}

__global__ void add_check_against_constants
(var *a, var *b, var *c, var *x, var *y, var *z, var *res_x, var *res_y, var *res_z) {
    g1_gpu::element lhs;
    g1_gpu::element rhs;
    g1_gpu::element result;
    g1_gpu::element expected;

    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        lhs.x.data[tid] = fq_gpu::to_monty(a[tid], res_x[tid]);
        lhs.y.data[tid] = fq_gpu::to_monty(b[tid], res_x[tid]);
        lhs.z.data[tid] = fq_gpu::to_monty(c[tid], res_x[tid]);
        rhs.x.data[tid] = fq_gpu::to_monty(x[tid], res_x[tid]);
        rhs.y.data[tid] = fq_gpu::to_monty(y[tid], res_x[tid]);
        rhs.z.data[tid] = fq_gpu::to_monty(z[tid], res_x[tid]);

        // lhs + rhs (affine element + affine element)
        g1_gpu::add(
            lhs.x.data[tid], lhs.y.data[tid], lhs.z.data[tid], 
            rhs.x.data[tid], rhs.y.data[tid], rhs.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );
        
        // Transform results from montgomery form 
        fq_gpu::from_monty(res_x[tid], res_x[tid]);
        fq_gpu::from_monty(res_y[tid], res_y[tid]);
        fq_gpu::from_monty(res_z[tid], res_z[tid]);
    }
}

/* -------------------------- Add Exception Test ---------------------------------------------- */

__global__ void initialize_add_exception_test_dbl
(var *a, var *b, var *c, var *x, var *y, var *z) {
    fq_gpu a_x{ 0x184b38afc6e2e09a, 0x4965cd1c3687f635, 0x334da8e7539e71c4, 0xf708d16cfe6e14 };
    fq_gpu a_y{ 0x2a6ff6ffc739b3b6, 0x70761d618b513b9, 0xbf1645401de26ba1, 0x114a1616c164b980 };
    fq_gpu a_z{ 0x10143ade26bbd57a, 0x98cf4e1f6c214053, 0x6bfdc534f6b00006, 0x1875e5068ababf2c };
    fq_gpu b_x{ 0x184b38afc6e2e09a, 0x4965cd1c3687f635, 0x334da8e7539e71c4, 0xf708d16cfe6e14 };
    fq_gpu b_y{ 0x2a6ff6ffc739b3b6, 0x70761d618b513b9, 0xbf1645401de26ba1, 0x114a1616c164b980 };
    fq_gpu b_z{ 0x10143ade26bbd57a, 0x98cf4e1f6c214053, 0x6bfdc534f6b00006, 0x1875e5068ababf2c };

    for (int i = 0; i < LIMBS_NUM; i++) {
        a[i] = a_x.data[i];
        b[i] = a_y.data[i];
        c[i] = a_z.data[i];
        x[i] = b_x.data[i];
        y[i] = b_y.data[i];
        z[i] = b_z.data[i];
    }
}

__global__ void add_exception_test_dbl
(var *a, var *b, var *c, var *x, var *y, var *z, var *expected_x, var *expected_y, var *expected_z, var *res_x, var *res_y, var *res_z) {
    g1_gpu::element lhs;
    g1_gpu::element rhs;
    g1_gpu::element result;
    g1_gpu::element expected;

    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        lhs.x.data[tid] = fq_gpu::load(a[tid], expected_x[tid]);
        lhs.y.data[tid] = fq_gpu::load(b[tid], expected_x[tid]);
        lhs.z.data[tid] = fq_gpu::load(c[tid], expected_x[tid]);
        rhs.x.data[tid] = fq_gpu::load(x[tid], expected_x[tid]);
        rhs.y.data[tid] = fq_gpu::load(y[tid], expected_x[tid]);
        rhs.z.data[tid] = fq_gpu::load(z[tid], expected_x[tid]);

        // lhs + rhs
        g1_gpu::add(
            lhs.x.data[tid], lhs.y.data[tid], lhs.z.data[tid], 
            rhs.x.data[tid], rhs.y.data[tid], rhs.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // Temporarily handle case where P = Q -- NEED TO MOVE TO 'group.cu' file
        if (fq_gpu::is_zero(res_x[tid]) && fq_gpu::is_zero(res_y[tid]) && fq_gpu::is_zero(res_z[tid])) {
            g1_gpu::doubling(
                lhs.x.data[tid], lhs.y.data[tid], lhs.z.data[tid], 
                res_x[tid], res_y[tid], res_z[tid]
            );
        }

        // lhs.doubling
        g1_gpu::doubling(
            lhs.x.data[tid], lhs.y.data[tid], lhs.z.data[tid], 
            expected_x[tid], expected_y[tid], expected_z[tid]
        );

        // Transform results from montgomery form 
        fq_gpu::from_monty(res_x[tid], res_x[tid]);
        fq_gpu::from_monty(res_y[tid], res_y[tid]);
        fq_gpu::from_monty(res_z[tid], res_z[tid]);

        // Transform results from montgomery form 
        fq_gpu::from_monty(expected_x[tid], expected_x[tid]);
        fq_gpu::from_monty(expected_y[tid], expected_y[tid]);
        fq_gpu::from_monty(expected_z[tid], expected_z[tid]);

        // EXPECT(lsh + rhs == lhs.doubling);
    }
}

/* -------------------------- Add Double Consistency Test ---------------------------------------------- */

__global__ void initialize_add_dbl_consistency
(var *a, var *b, var *c, var *x, var *y, var *z) {
    fq_gpu a_x{ 0x184b38afc6e2e09a, 0x4965cd1c3687f635, 0x334da8e7539e71c4, 0xf708d16cfe6e14 };
    fq_gpu a_y{ 0x2a6ff6ffc739b3b6, 0x70761d618b513b9, 0xbf1645401de26ba1, 0x114a1616c164b980 };
    fq_gpu a_z{ 0x10143ade26bbd57a, 0x98cf4e1f6c214053, 0x6bfdc534f6b00006, 0x1875e5068ababf2c };
    fq_gpu b_x{ 0x184b38afc6e2e09a, 0x4965cd1c3687f635, 0x334da8e7539e71c4, 0xf708d16cfe6e14 };
    fq_gpu b_y{ 0x2a6ff6ffc739b3b6, 0x70761d618b513b9, 0xbf1645401de26ba1, 0x114a1616c164b980 };
    fq_gpu b_z{ 0x10143ade26bbd57a, 0x98cf4e1f6c214053, 0x6bfdc534f6b00006, 0x1875e5068ababf2c };

    for (int i = 0; i < LIMBS_NUM; i++) {
        a[i] = a_x.data[i];
        b[i] = a_y.data[i];
        c[i] = a_z.data[i];
        x[i] = b_x.data[i];
        y[i] = b_y.data[i];
        z[i] = b_z.data[i];
    }
}

__global__ void add_dbl_consistency
(var *a, var *b, var *c, var *x, var *y, var *z, var *expected_x, var *expected_y, var *expected_z, var *res_x, var *res_y, var *res_z) {
    g1_gpu::element a_element;
    g1_gpu::element b_element;
    g1_gpu::element c_element;
    g1_gpu::element d_element;
    g1_gpu::element add_result;
    g1_gpu::element dbl_result;

    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        a_element.x.data[tid] = fq_gpu::load(a[tid], res_x[tid]);
        a_element.y.data[tid] = fq_gpu::load(b[tid], res_x[tid]);
        a_element.z.data[tid] = fq_gpu::load(c[tid], res_x[tid]);
        b_element.x.data[tid] = fq_gpu::load(x[tid], res_x[tid]);
        b_element.y.data[tid] = fq_gpu::load(y[tid], res_x[tid]);
        b_element.z.data[tid] = fq_gpu::load(z[tid], res_x[tid]);

        // c = a + b
        g1_gpu::add(
            a_element.x.data[tid], a_element.y.data[tid], a_element.z.data[tid], 
            b_element.x.data[tid], b_element.y.data[tid], b_element.z.data[tid], 
            c_element.x.data[tid], c_element.y.data[tid], c_element.z.data[tid]
        ); 
        
        // b = -b
        fq_gpu::neg(b_element.y.data[tid], b_element.y.data[tid]);                                                                                                                                                      
        
        // d = a + b
        g1_gpu::add(
            a_element.x.data[tid], a_element.y.data[tid], a_element.z.data[tid], 
            b_element.x.data[tid], b_element.y.data[tid], b_element.z.data[tid], 
            d_element.x.data[tid], d_element.y.data[tid], d_element.z.data[tid]
        );
       
        // result = c + d
        g1_gpu::add(
            c_element.x.data[tid], c_element.y.data[tid], c_element.z.data[tid], 
            d_element.x.data[tid], d_element.y.data[tid], d_element.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // Temporarily handle case where P = Q -- NEED TO MOVE TO 'group.cu' file
        if (fq_gpu::is_zero(res_x[tid]) && fq_gpu::is_zero(res_y[tid]) && fq_gpu::is_zero(res_z[tid])) {
            g1_gpu::doubling(
                a_element.x.data[tid], a_element.y.data[tid], a_element.z.data[tid], 
                res_x[tid], res_y[tid], res_z[tid]
            );
        }

        // a.doubling
        g1_gpu::doubling(
            a_element.x.data[tid], a_element.y.data[tid], a_element.z.data[tid], 
            expected_x[tid], expected_y[tid], expected_z[tid]
        );
         
        // Transform results from montgomery form 
        fq_gpu::from_monty(res_x[tid], res_x[tid]);
        fq_gpu::from_monty(res_y[tid], res_y[tid]);
        fq_gpu::from_monty(res_z[tid], res_z[tid]);
        
        // Transform results from montgomery form 
        fq_gpu::from_monty(expected_x[tid], expected_x[tid]);
        fq_gpu::from_monty(expected_y[tid], expected_y[tid]);
        fq_gpu::from_monty(expected_z[tid], expected_z[tid]);

        // EXPECT (c + d == a.doubling);
    }
}

/* -------------------------- Add Double Consistency Repeated Test ---------------------------------------------- */

__global__ void initialize_add_dbl_consistency_repeated
(var *a, var *b, var *c) {
    fq_gpu a_x{ 0x184b38afc6e2e09a, 0x4965cd1c3687f635, 0x334da8e7539e71c4, 0xf708d16cfe6e14 };
    fq_gpu a_y{ 0x2a6ff6ffc739b3b6, 0x70761d618b513b9, 0xbf1645401de26ba1, 0x114a1616c164b980 };
    fq_gpu a_z{ 0x10143ade26bbd57a, 0x98cf4e1f6c214053, 0x6bfdc534f6b00006, 0x1875e5068ababf2c };

    for (int i = 0; i < LIMBS_NUM; i++) {
        a[i] = a_x.data[i];
        b[i] = a_y.data[i];
        c[i] = a_z.data[i];
    }
}

__global__ void add_dbl_consistency_repeated
(var *a, var *b, var *c, var *expected_x, var *expected_y, var *expected_z, var *res_x, var *res_y, var *res_z) {
    g1_gpu::element a_element;
    g1_gpu::element b_element;
    g1_gpu::element c_element;
    g1_gpu::element d_element;
    g1_gpu::element e_element;
    g1_gpu::element result;
    g1_gpu::element expected;

    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        a_element.x.data[tid] = fq_gpu::load(a[tid], res_x[tid]);
        a_element.y.data[tid] = fq_gpu::load(b[tid], res_x[tid]);
        a_element.z.data[tid] = fq_gpu::load(c[tid], res_x[tid]);

        // b = 2a
        g1_gpu::doubling(
            a_element.x.data[tid], a_element.y.data[tid], a_element.z.data[tid], 
            b_element.x.data[tid], b_element.y.data[tid], b_element.z.data[tid]
        );

        // c = 4a
        g1_gpu::doubling(
            b_element.x.data[tid], b_element.y.data[tid], b_element.z.data[tid], 
            c_element.x.data[tid], c_element.y.data[tid], c_element.z.data[tid]
        );
         
        // d = 3a
        g1_gpu::add(
            a_element.x.data[tid], a_element.y.data[tid], a_element.z.data[tid], 
            b_element.x.data[tid], b_element.y.data[tid], b_element.z.data[tid], 
            d_element.x.data[tid], d_element.y.data[tid], d_element.z.data[tid]
        ); 

        // e = 5a
        g1_gpu::add(
            a_element.x.data[tid], a_element.y.data[tid], a_element.z.data[tid], 
            c_element.x.data[tid], c_element.y.data[tid], c_element.z.data[tid], 
            e_element.x.data[tid], e_element.y.data[tid], e_element.z.data[tid]
        ); 
  
        // result = 8a
        g1_gpu::add(
            d_element.x.data[tid], d_element.y.data[tid], d_element.z.data[tid], 
            e_element.x.data[tid], e_element.y.data[tid], e_element.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // c.doubling
        g1_gpu::doubling(
            c_element.x.data[tid], c_element.y.data[tid], c_element.z.data[tid], 
            expected_x[tid], expected_y[tid], expected_z[tid]
        );

        // Transform results from montgomery form 
        fq_gpu::from_monty(res_x[tid], res_x[tid]);
        fq_gpu::from_monty(res_y[tid], res_y[tid]);
        fq_gpu::from_monty(res_z[tid], res_z[tid]);

        // Transform results from montgomery form 
        fq_gpu::from_monty(expected_x[tid], expected_x[tid]);
        fq_gpu::from_monty(expected_y[tid], expected_y[tid]);
        fq_gpu::from_monty(expected_z[tid], expected_z[tid]);

        // EXPECT (d + e == c.doubling)
    }
}

/* -------------------------- Group Exponentiation Check Against Constants Test ---------------------------------------------- */

__global__ void initialize_group_exponentiation
(var *a, var *b, var *c, var *expected_x, var *expected_y, var *expected_z) {
    fq_gpu a_x{ 0x184b38afc6e2e09a, 0x4965cd1c3687f635, 0x334da8e7539e71c4, 0xf708d16cfe6e14 };
    fq_gpu a_y{ 0x2a6ff6ffc739b3b6, 0x70761d618b513b9, 0xbf1645401de26ba1, 0x114a1616c164b980 };
    fq_gpu a_z{ 0x10143ade26bbd57a, 0x98cf4e1f6c214053, 0x6bfdc534f6b00006, 0x1875e5068ababf2c };
    fq_gpu exp_x{ 0xC22BA855EE138794, 0xA61591A7E7FD82BF, 0xE156E7E491B4B7E2, 0x2F4620C8373C106A };
    fq_gpu exp_y{ 0xFAFBA721679C418, 0xE5491810D637BB55, 0x64B6FAD0A59D97B2, 0x111DA26AEEE41706 };
    fq_gpu exp_z{ 0x59F11DAE3A07BF31, 0xDB2756DB66333FB, 0x34F2D97DAD44161, 0xD1A485A89C277DA };

    for (int i = 0; i < LIMBS_NUM; i++) {
        a[i] = a_x.data[i];
        b[i] = a_y.data[i];
        c[i] = a_z.data[i];
        expected_x[i] = exp_x.data[i];
        expected_y[i] = exp_y.data[i];
        expected_z[i] = exp_z.data[i];
    }
}

__global__ void group_exponentiation(uint64_t *a, uint64_t *b, uint64_t *c, var *res_x, var *res_y, var *res_z) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    g1_gpu::element one; 
    g1_gpu::element R;
    g1_gpu::element Q;

    fr_gpu exponent{ 0xb67299b792199cf0, 0xc1da7df1e7e12768, 0x692e427911532edf, 0x13dd85e87dc89978 };

    fq_gpu::load(gpu_barretenberg::one_x_bn_254[tid], one.x.data[tid]);
    fq_gpu::load(gpu_barretenberg::one_y_bn_254[tid], one.y.data[tid]);
    fq_gpu::load(fq_gpu::one().data[tid], one.z.data[tid]);

    if (tid < LIMBS) {
        // Initialize 'R' to the identity element, Q to the curve point
        fq_gpu::load(0, R.x.data[tid]); 
        fq_gpu::load(0, R.y.data[tid]); 
        fq_gpu::load(0, R.z.data[tid]); 

        fq_gpu::load(one.x.data[tid], Q.x.data[tid]);
        fq_gpu::load(one.y.data[tid], Q.y.data[tid]);
        fq_gpu::load(one.z.data[tid], Q.z.data[tid]);

        // Loop for each limb starting with the last limb
        for (int j = 3; j >= 0; j--) {
            // Loop for each bit of scalar
            for (int i = 64; i >= 0; i--) {
                // Performs bit-decompositon by traversing the bits of the scalar from MSB to LSB
                // and extracting the i-th bit of scalar in limb.
                if (((exponent.data[j] >> i) & 1) ? 1 : 0)
                    g1_gpu::add(
                        R.x.data[tid], R.y.data[tid], R.z.data[tid], 
                        Q.x.data[tid], Q.y.data[tid], Q.z.data[tid], 
                        R.x.data[tid], R.y.data[tid], R.z.data[tid]
                    );
                if (i != 0) 
                    g1_gpu::doubling(
                        R.x.data[tid], R.y.data[tid], R.z.data[tid], 
                        R.x.data[tid], R.y.data[tid], R.z.data[tid]
                    );
            }
        }
    }

    // Store the final value of R into the result array for this limb
    fq_gpu::load(R.x.data[tid], res_x[tid]);
    fq_gpu::load(R.y.data[tid], res_y[tid]);
    fq_gpu::load(R.z.data[tid], res_z[tid]);

    // Convert back from montgomery form
    fq_gpu::from_monty(res_x[tid], res_x[tid]);
    fq_gpu::from_monty(res_y[tid], res_y[tid]);
    fq_gpu::from_monty(res_z[tid], res_z[tid]);
}

/* -------------------------- Operator Ordering Test ---------------------------------------------- */

__global__ void initialize_operator_ordering(var *a, var *b, var *c, var *d) {
    fr_gpu scalar{ 0xcfbfd4441138823e, 0xb5f817e28a1ef904, 0xefb7c5629dcc1c42, 0x1a9ed3d6f846230e };

    for (int i = 0; i < LIMBS_NUM; i++) {
        a[i] = gpu_barretenberg::one_x_bn_254[i];
        b[i] = gpu_barretenberg::one_y_bn_254[i];
        c[i] = fq_gpu::one().data[i];
        d[i] = scalar.data[i];
    }
}

__global__ void operator_ordering(uint64_t *a, uint64_t *b, uint64_t *c, uint64_t *d, var *res_x, var *res_y, var *res_z) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    g1_gpu::element b_new;
    g1_gpu::element c_new;
    g1_gpu::element d_new;

    // Copy a into b
    fq_gpu::load(a[tid], b_new.x.data[tid]);
    fq_gpu::load(b[tid], b_new.y.data[tid]);
    fq_gpu::load(c[tid], b_new.z.data[tid]);

    // c = a + b
    g1_gpu::add(
        a[tid], b[tid], c[tid], 
        b_new.x.data[tid], b_new.y.data[tid], b_new.z.data[tid], 
        c_new.x.data[tid], c_new.y.data[tid], c_new.z.data[tid]
    );

    // Double is c == 0
    if (fq_gpu::is_zero(c_new.x.data[tid]) && fq_gpu::is_zero(c_new.y.data[tid]) && fq_gpu::is_zero(c_new.z.data[tid])) {
        g1_gpu::doubling(
            a[tid], b[tid], c[tid], 
            c_new.x.data[tid], c_new.y.data[tid], c_new.z.data[tid]
        );
    }

    // d = b + a
    g1_gpu::add(
        b_new.x.data[tid], b_new.y.data[tid], b_new.z.data[tid], 
        a[tid], b[tid], c[tid], 
        d_new.x.data[tid], d_new.y.data[tid], d_new.z.data[tid]
    );

    // Double is d == 0
    if (fq_gpu::is_zero(d_new.x.data[tid]) && fq_gpu::is_zero(d_new.y.data[tid]) && fq_gpu::is_zero(d_new.z.data[tid])) {
        g1_gpu::doubling(
            b_new.x.data[tid], b_new.y.data[tid], b_new.z.data[tid], 
            d_new.x.data[tid], d_new.y.data[tid], d_new.z.data[tid]
        );
    }

    // Return final result. Expect c == d
    fq_gpu::load(d_new.x.data[tid], res_x[tid]);
    fq_gpu::load(d_new.y.data[tid], res_y[tid]);
    fq_gpu::load(d_new.z.data[tid], res_z[tid]);

    fq_gpu::from_monty(res_x[tid], res_x[tid]);
    fq_gpu::from_monty(res_y[tid], res_y[tid]);
    fq_gpu::from_monty(res_z[tid], res_z[tid]);
}

/* -------------------------- Executing Initialization and Workload Kernels ---------------------------------------------- */

void assert_checks(var *expected, var *result) {
    // Explicit synchronization barrier
    hipDeviceSynchronize();
    
    // Assert clause
    assert(expected[0] == result[0]);
    assert(expected[1] == result[1]);
    assert(expected[2] == result[2]);
    assert(expected[3] == result[3]);

    // Print statements
    printf("expected[0] is: %zu\n", expected[0]);
    printf("expected[1] is: %zu\n", expected[1]);
    printf("expected[2] is: %zu\n", expected[2]);
    printf("expected[3] is: %zu\n", expected[3]);
    printf("result[0] is: %zu\n", result[0]);
    printf("result[1] is: %zu\n", result[1]);
    printf("result[2] is: %zu\n", result[2]);
    printf("result[3] is: %zu\n", result[3]);
}

void execute_kernels
(var *a, var *b, var *c, var *x, var *y, var *z, var *expected_x, var *expected_y, var *expected_z, var *res_x, var *res_y, var *res_z) {
    // Mixed Addition Test
    initialize_mixed_add_check_against_constants<<<BLOCKS, THREADS>>>(a, b, c, x, y, z, expected_x, expected_y, expected_z);
    mixed_add_check_against_constants<<<BLOCKS, LIMBS_NUM>>>(a, b, c, x, y, z, res_x, res_y, res_z);
    assert_checks(expected_x, res_x);
    assert_checks(expected_y, res_y);
    assert_checks(expected_z, res_z);

    // Doubling Test
    initialize_dbl_check_against_constants<<<BLOCKS, THREADS>>>(a, b, c, x, y, z, expected_x, expected_y, expected_z);
    dbl_check_against_constants<<<BLOCKS, LIMBS_NUM>>>(a, b, c, x, y, z, res_x, res_y, res_z);
    assert_checks(expected_x, res_x);
    assert_checks(expected_y, res_y);
    assert_checks(expected_z, res_z);

    // Addition Test
    initialize_add_check_against_constants<<<BLOCKS, THREADS>>>(a, b, c, x, y, z, expected_x, expected_y, expected_z);
    add_check_against_constants<<<BLOCKS, LIMBS_NUM>>>(a, b, c, x, y, z, res_x, res_y, res_z);
    assert_checks(expected_x, res_x);
    assert_checks(expected_y, res_y);
    assert_checks(expected_z, res_z);

    // Add Exception Test
    initialize_add_exception_test_dbl<<<BLOCKS, THREADS>>>(a, b, c, x, y, z);
    add_exception_test_dbl<<<BLOCKS, LIMBS_NUM>>>(a, b, c, x, y, z, expected_x, expected_y, expected_z, res_x, res_y, res_z);
    assert_checks(expected_x, res_x);
    assert_checks(expected_y, res_y);
    assert_checks(expected_z, res_z);

    // Add Double Consistency Test
    initialize_add_dbl_consistency<<<BLOCKS, THREADS>>>(a, b, c, x, y, z);
    add_dbl_consistency<<<BLOCKS, LIMBS_NUM>>>(a, b, c, x, y, z, expected_x, expected_y, expected_z, res_x, res_y, res_z);
    assert_checks(expected_x, res_x);
    assert_checks(expected_y, res_y);
    assert_checks(expected_z, res_z);

    // Add Double Consistency Repeated Test
    initialize_add_dbl_consistency_repeated<<<BLOCKS, THREADS>>>(a, b, c);
    add_dbl_consistency_repeated<<<BLOCKS, LIMBS_NUM>>>(a, b, c, expected_x, expected_y, expected_z, res_x, res_y, res_z);

    // Group Exponentiation Consistency Test
    initialize_group_exponentiation<<<BLOCKS, THREADS>>>(a, b, c, expected_x, expected_y, expected_z);
    group_exponentiation<<<BLOCKS, LIMBS_NUM>>>(a, b, c, res_x, res_y, res_z);
    assert_checks(expected_x, res_x);
    assert_checks(expected_y, res_y);
    assert_checks(expected_z, res_z);

    // Operator Ordering Test
    initialize_operator_ordering<<<BLOCKS, THREADS>>>(a, b, c, x);
    operator_ordering<<<BLOCKS, LIMBS_NUM>>>(a, b, c, x, res_x, res_y, res_z);
    assert_checks(expected_x, res_x);
    assert_checks(expected_y, res_y);
    assert_checks(expected_z, res_z);
}

/* -------------------------- Main Entry Function ---------------------------------------------- */

int main(int, char**) {
    // Start timer
    auto start = high_resolution_clock::now();

    // Define pointers to 'uint64_t' type
    var *a, *b, *c, *x, *y, *z, *expected_x, *expected_y, *expected_z, *res_x, *res_y, *res_z;    

    // Allocate unified memory accessible by host and device
    hipMallocManaged(&a, LIMBS_NUM * sizeof(uint64_t));
    hipMallocManaged(&b, LIMBS_NUM * sizeof(uint64_t));
    hipMallocManaged(&c, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&x, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&y, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&z, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&expected_x, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&expected_y, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&expected_z, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&res_x, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&res_y, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&res_z, LIMBS * sizeof(uint64_t));

    // Execute kernel functions
    execute_kernels(a, b, c, x, y, z, expected_x, expected_y, expected_z, res_x, res_y, res_z);

    // Successfull execution of unit tests
    cout << "******* All 'g1_gpu BN-254 Curve' unit tests passed! **********" << endl;

    // End timer
    auto stop = high_resolution_clock::now();

    // Calculate duraion of execution time 
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time taken by function: " << duration.count() << " microseconds\n" << endl; 

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(x);
    hipFree(y);
    hipFree(z);
    hipFree(expected_x);
    hipFree(expected_y);
    hipFree(expected_z);
    hipFree(res_x);
    hipFree(res_y);
    hipFree(res_z);

    cout << "Completed sucessfully!" << endl;

    return 0;
}
