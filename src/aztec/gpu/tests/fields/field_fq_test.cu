#include "hip/hip_runtime.h"
#include "field.cu"
#include <assert.h> 

using namespace std;
using namespace std::chrono;
using namespace gpu_barretenberg;

static constexpr size_t LIMBS_NUM = 4;
static constexpr size_t BLOCKS = 1;
static constexpr size_t THREADS = 1;

/* -------------------------- Montgomery Multiplication Test ---------------------------------------------- */

__global__ void initialize_mont_mult(uint64_t *a, uint64_t *b, uint64_t *expected) {
    fq_gpu a_field{ 0x2523b6fa3956f038, 0x158aa08ecdd9ec1d, 0xf48216a4c74738d4, 0x2514cc93d6f0a1bf };
    fq_gpu b_field{ 0xb68aee5e4c8fc17c, 0xc5193de7f401d5e8, 0xb8777d4dde671db3, 0xe513e75c087b0bb };
    fq_gpu expect{ 0x7ed4174114b521c4, 0x58f5bd1d4279fdc2, 0x6a73ac09ee843d41, 0x687a76ae9b3425c };

    for (int i = 0; i < LIMBS_NUM; i++) {
        a[i] = a_field.data[i];
        b[i] = b_field.data[i];
        expected[i] = expect.data[i];
    }
}

__global__ void mont_mult(uint64_t *a, uint64_t *b, uint64_t *result) {
    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        fq_gpu::mul(a[tid], b[tid], result[tid]); 
    }
}

/* -------------------------- Montgomery Multiplication Test -- Short Integers ---------------------------------------------- */

__global__ void initialize_mont_mult_short(uint64_t *a, uint64_t *b, uint64_t *expected) {
    // fq_gpu a_field{ 0xa, 0, 0, 0 };
    // fq_gpu b_field{ 0xb, 0, 0, 0 };
    fq_gpu a_field{ 0x1, 0, 0, 0 };
    fq_gpu b_field{ 0x1, 0, 0, 0 };
    fq_gpu expect = { 0x65991a6dc2f3a183, 0xe3ba1f83394a2d08, 0x8401df65a169db3f, 0x1727099643607bba };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = a_field.data[i];
        b[i] = b_field.data[i];
        expected[i] = expect.data[i];
    }
}

__global__ void mont_mult_short(uint64_t *a, uint64_t *b, uint64_t *result) {
    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        fq_gpu::mul(a[tid], b[tid], result[tid]); 
    }
}

/* -------------------------- Multiply - Square Consistency ---------------------------------------------- */

__global__ void initialize_mul_square_consistency(uint64_t *a, uint64_t *b) {
    fq_gpu a_field{ 0x329596aa978981e8, 0x8542e6e254c2a5d0, 0xc5b687d82eadb178, 0x2d242aaf48f56b8a };
    fq_gpu b_field{ 0x7d2e20e82f73d3e8, 0x8e50616a7a9d419d, 0xcdc833531508914b, 0xd510253a2ce62c };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = a_field.data[i];
        b[i] = b_field.data[i];
    }
}

__global__ void mul_square_consistency(uint64_t *a, uint64_t *b, uint64_t *expected, uint64_t *result) {
    fq_gpu t1;
    fq_gpu t2;

    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        t1.data[tid] = fq_gpu::sub(a[tid], b[tid], result[tid]);
        t2.data[tid] = fq_gpu::add(a[tid], b[tid], result[tid]);
        fq_gpu::mul(t1.data[tid], t2.data[tid], expected[tid]);

        t1.data[tid] = fq_gpu::square(a[tid], result[tid]);
        t2.data[tid] = fq_gpu::square(b[tid], result[tid]);
        fq_gpu::sub(t1.data[tid], t2.data[tid], result[tid]);
    }
}

/* -------------------------- Multiply - Square Against Constants ---------------------------------------------- */

__global__ void initialize_sqr_check_against_constants(uint64_t *a, uint64_t *expected) {
    fq_gpu a_field{ 0x329596aa978981e8, 0x8542e6e254c2a5d0, 0xc5b687d82eadb178, 0x2d242aaf48f56b8a };
    fq_gpu expect = { 0xbf4fb34e120b8b12, 0xf64d70efbf848328, 0xefbb6a533f2e7d89, 0x1de50f941425e4aa };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = a_field.data[i];
        expected[i] = expect.data[i];
    }
}

__global__ void sqr_check_against_constants(uint64_t *a, uint64_t *result) {
    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        fq_gpu::square(a[tid], result[tid]);
    }
}

/* -------------------------- Add - Check Against Constants ---------------------------------------------- */

__global__ void initialize_add_check_against_constants(uint64_t *a, uint64_t *b, uint64_t *expected) {
    fq_gpu a_field{ 0x7d2e20e82f73d3e8, 0x8e50616a7a9d419d, 0xcdc833531508914b, 0xd510253a2ce62c };
    fq_gpu b_field{ 0x2829438b071fd14e, 0xb03ef3f9ff9274e, 0x605b671f6dc7b209, 0x8701f9d971fbc9 };
    fq_gpu expect{ 0xa55764733693a536, 0x995450aa1a9668eb, 0x2e239a7282d04354, 0x15c121f139ee1f6 };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = a_field.data[i];
        b[i] = b_field.data[i];
        expected[i] = expect.data[i];
    }
}

__global__ void add_check_against_constants(uint64_t *a, uint64_t *b, uint64_t *result) {
    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        fq_gpu::add(a[tid], b[tid], result[tid]);
    }
}

/* -------------------------- Subtract - Check Against Constants ---------------------------------------------- */

__global__ void initialize_sub_check_against_constants(uint64_t *a, uint64_t *b, uint64_t *expected) {
    fq_gpu a_field{ 0xd68d01812313fb7c, 0x2965d7ae7c6070a5, 0x08ef9af6d6ba9a48, 0x0cb8fe2108914f53 };
    fq_gpu b_field{ 0x2cd2a2a37e9bf14a, 0xebc86ef589c530f6, 0x75124885b362b8fe, 0x1394324205c7a41d };
    fq_gpu expect{ 0xe5daeaf47cf50779, 0xd51ed34a5b0d0a3c, 0x4c2d9827a4d939a6, 0x29891a51e3fb4b5f };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = a_field.data[i];
        b[i] = b_field.data[i];
        expected[i] = expect.data[i];
    }
}

__global__ void sub_check_against_constants(uint64_t *a, uint64_t *b, uint64_t *result) {
    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        fq_gpu::sub(a[tid], b[tid], result[tid]);
    }
}

/* -------------------------- Convert To Montgomery Form ---------------------------------------------- */

__global__ void initialize_to_montgomery_form(uint64_t *a, uint64_t *expected) {
    fq_gpu a_field{ 0x01, 0x00, 0x00, 0x00 };
    fq_gpu expect{ 0xd35d438dc58f0d9d, 0xa78eb28f5c70b3d, 0x666ea36f7879462c, 0xe0a77c19a07df2f };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = a_field.data[i];
        expected[i] = expect.data[i];
    }
}

__global__ void to_montgomery_form(uint64_t *a, uint64_t *result) {
    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        fq_gpu::to_monty(a[tid], result[tid]);
    }
}

/* -------------------------- Convert From Montgomery Form ---------------------------------------------- */

__global__ void initialize_from_montgomery_form(uint64_t *a, uint64_t *expected) {
    fq_gpu a_field{ 0x01, 0x00, 0x00, 0x00 };
    fq_gpu expect{ 0x01, 0x00, 0x00, 0x00 };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = a_field.data[i];
        expected[i] = expect.data[i];        
    }
}

__global__ void from_montgomery_form(uint64_t *a, uint64_t *result) {
    fq_gpu t1;

    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        t1.data[tid] = fq_gpu::to_monty(a[tid], result[tid]);
        fq_gpu::from_monty(t1.data[tid], result[tid]);
    }
}

/* -------------------------- Montgomery Consistency Check ---------------------------------------------- */

__global__ void initialize_montgomery_consistency_check(uint64_t *a, uint64_t *b) {
    fq_gpu a_field{ 0x2523b6fa3956f038, 0x158aa08ecdd9ec1d, 0xf48216a4c74738d4, 0x2514cc93d6f0a1bf };
    fq_gpu b_field{ 0xb68aee5e4c8fc17c, 0xc5193de7f401d5e8, 0xb8777d4dde671db3, 0xe513e75c087b0bb };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = a_field.data[i];
        b[i] = b_field.data[i];
    }
}

__global__ void montgomery_consistency_check(uint64_t *a, uint64_t *b, uint64_t *expected, uint64_t *result) {
    fq_gpu aR;
    fq_gpu bR;
    fq_gpu aRR;
    fq_gpu bRR;
    fq_gpu bRRR;
    fq_gpu result_a;
    fq_gpu result_b;
    fq_gpu result_c;
    fq_gpu result_d;

    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        aR.data[tid] = fq_gpu::to_monty(a[tid], result[tid]);
        aRR.data[tid] = fq_gpu::to_monty(aR.data[tid], result[tid]);
        bR.data[tid] = fq_gpu::to_monty(b[tid], result[tid]);
        bRR.data[tid] = fq_gpu::to_monty(bR.data[tid], result[tid]);
        bRRR.data[tid] = fq_gpu::to_monty(bRR.data[tid], result[tid]);

        result_a.data[tid] = fq_gpu::mul(aRR.data[tid], bRR.data[tid], result[tid]); // abRRR
        result_b.data[tid] = fq_gpu::mul(aR.data[tid], bRRR.data[tid], result[tid]); // abRRR
        result_c.data[tid] = fq_gpu::mul(aR.data[tid], bR.data[tid], result[tid]);   // abR
        result_d.data[tid] = fq_gpu::mul(a[tid], b[tid], result[tid]);               // abR^-1

        result_a.data[tid] = fq_gpu::from_monty(result_a.data[tid], result[tid]);    // abRR
        result_a.data[tid] = fq_gpu::from_monty(result_a.data[tid], result[tid]);    // abR
        result_a.data[tid] = fq_gpu::from_monty(result_a.data[tid], result[tid]);    // ab
        result_c.data[tid] = fq_gpu::from_monty(result_c.data[tid], expected[tid]);    // ab
        result_d.data[tid] = fq_gpu::to_monty(result_d.data[tid], expected[tid]);      // ab
    }
}

/* -------------------------- Add Multiplication Consistency ---------------------------------------------- */

__global__ void initialize_add_mul_consistency(uint64_t *a, uint64_t *b) {
    fq_gpu a_field{ 0x2523b6fa3956f038, 0x158aa08ecdd9ec1d, 0xf48216a4c74738d4, 0x2514cc93d6f0a1bf };
    fq_gpu multiplicand = { 0x09, 0, 0, 0 };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = a_field.data[i];
        b[i] = multiplicand.data[i];
    }
}

__global__ void add_mul_consistency(uint64_t *a, uint64_t *b, uint64_t *expected, uint64_t *res) {
    fq_gpu multiplicand;
    fq_gpu result;

    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        multiplicand.data[tid] = fq_gpu::to_monty(b[tid], res[tid]);    
        result.data[tid] = fq_gpu::add(a[tid], a[tid], res[tid]);                       // 2
        result.data[tid] = fq_gpu::add(result.data[tid], result.data[tid], res[tid]);   // 4
        result.data[tid] = fq_gpu::add(result.data[tid], result.data[tid], res[tid]);   // 8
        result.data[tid] = fq_gpu::add(result.data[tid], a[tid], res[tid]);             // 9

        fq_gpu::mul(a[tid], multiplicand.data[tid], expected[tid]);                     // 9        
    }
}

/* -------------------------- Subtract Multiplication Consistency ---------------------------------------------- */

__global__ void initialize_sub_mul_consistency(uint64_t *a, uint64_t *b) {
    fq_gpu a_field{ 0x2523b6fa3956f038, 0x158aa08ecdd9ec1d, 0xf48216a4c74738d4, 0x2514cc93d6f0a1bf };
    fq_gpu multiplicand = { 0x05, 0, 0, 0 };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = a_field.data[i];
        b[i] = multiplicand.data[i];
    }
}

__global__ void sub_mul_consistency(uint64_t *a, uint64_t *b, uint64_t *expected, uint64_t *res) {
    fq_gpu multiplicand;
    fq_gpu result;

    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        multiplicand.data[tid] = fq_gpu::to_monty(b[tid], res[tid]);    
        result.data[tid] = fq_gpu::add(a[tid], a[tid], res[tid]);                           // 2
        result.data[tid] = fq_gpu::add(result.data[tid], result.data[tid], res[tid]);       // 4
        result.data[tid] = fq_gpu::add(result.data[tid], result.data[tid], res[tid]);       // 8
        result.data[tid] = fq_gpu::sub(result.data[tid], a[tid], res[tid]);                 // 7
        result.data[tid] = fq_gpu::sub(result.data[tid], a[tid], res[tid]);                 // 6
        result.data[tid] = fq_gpu::sub(result.data[tid], a[tid], res[tid]);                 // 5

        fq_gpu::mul(a[tid], multiplicand.data[tid], expected[tid]);                         // 5       
    }
}

/* -------------------------- Cube Root ---------------------------------------------- */

__global__ void initialize_cube(uint64_t *a) {
    fq_gpu a_field{ 0x2523b6fa3956f038, 0x158aa08ecdd9ec1d, 0xf48216a4c74738d4, 0x2514cc93d6f0a1bf };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = a_field.data[i];
    }
}

__global__ void cube(uint64_t *a, uint64_t *expected, uint64_t *result) {
    fq_gpu x_cubed;
    fq_gpu beta_x;
    fq_gpu beta_x_cubed;

    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        x_cubed.data[tid] = fq_gpu::mul(a[tid], a[tid], result[tid]);  
        x_cubed.data[tid] = fq_gpu::mul(x_cubed.data[tid], a[tid], result[tid]);  

        beta_x.data[tid] = fq_gpu::mul(a[tid], gpu_barretenberg::CUBE_ROOT_BASE[tid], expected[tid]);  
        beta_x_cubed.data[tid] = fq_gpu::mul(beta_x.data[tid], beta_x.data[tid], expected[tid]); 
        beta_x_cubed.data[tid] = fq_gpu::mul(beta_x_cubed.data[tid], beta_x.data[tid], expected[tid]); 
    }
}

/* -------------------------- Executing Initialization and Workload Kernels ---------------------------------------------- */

void assert_checks(var *expected, var *result) {
    // Explicit synchronization barrier
    hipDeviceSynchronize();

    // Print statements
    printf("expected[0] is: %zx\n", expected[0]);
    printf("expected[1] is: %zx\n", expected[1]);
    printf("expected[2] is: %zx\n", expected[2]);
    printf("expected[3] is: %zx\n", expected[3]);
    printf("result[0] is: %zx\n", result[0]);
    printf("result[1] is: %zx\n", result[1]);
    printf("result[2] is: %zx\n", result[2]);
    printf("result[3] is: %zx\n", result[3]);

    // Assert clause
    if (expected[0] != result[0]) {
        printf("    BAD!!\n");
    }
    if (expected[1] != result[1]) {
        printf("    BAD!!\n");
    }
    if (expected[2] != result[2]) {
        printf("    BAD!!\n");
    }
    if (expected[3] != result[3]) {
        printf("    BAD!!\n");
    }
}

void execute_kernels(var *a, var *b, var *expected, var *result) {    
    // Montgomery Multiplication Test 
    initialize_mont_mult<<<BLOCKS, THREADS>>>(a, b, expected);
    hipDeviceSynchronize();
    mont_mult<<<BLOCKS, LIMBS_NUM>>>(a, b, result);
    assert_checks(expected, result);

    // Montgomery Multiplication Test -- Short Integers 
    initialize_mont_mult_short<<<BLOCKS, THREADS>>>(a, b, expected);
    hipDeviceSynchronize();
    mont_mult_short<<<BLOCKS, LIMBS_NUM>>>(a, b, result);
    assert_checks(expected, result);

    // Multiply Test - Square Consistency 
    initialize_mul_square_consistency<<<BLOCKS, THREADS>>>(a, b);
    hipDeviceSynchronize();
    mul_square_consistency<<<BLOCKS, LIMBS_NUM>>>(a, b, expected, result);
    assert_checks(expected, result);

    // Multiply Test - Square Against Constants 
    initialize_sqr_check_against_constants<<<BLOCKS, THREADS>>>(a, expected);
    hipDeviceSynchronize();
    sqr_check_against_constants<<<BLOCKS, LIMBS_NUM>>>(a, result);
    assert_checks(expected, result);

    // Add Test - Check Against Constants
    initialize_add_check_against_constants<<<BLOCKS, THREADS>>>(a, b, expected);
    hipDeviceSynchronize();
    add_check_against_constants<<<BLOCKS, LIMBS_NUM>>>(a, b, result);
    assert_checks(expected, result);

    // Subtract Test - Check Against Constant
    initialize_sub_check_against_constants<<<BLOCKS, THREADS>>>(a, b, expected);
    hipDeviceSynchronize();
    sub_check_against_constants<<<BLOCKS, LIMBS_NUM>>>(a, b, result);
    assert_checks(expected, result);

    // Convert To Montgomery Form Test
    initialize_to_montgomery_form<<<BLOCKS, THREADS>>>(a, expected);
    hipDeviceSynchronize();
    to_montgomery_form<<<BLOCKS, LIMBS_NUM>>>(a, result);
    assert_checks(expected, result);

    // Convert From Montgomery Form Test
    initialize_from_montgomery_form<<<BLOCKS, THREADS>>>(a, expected);
    hipDeviceSynchronize();
    from_montgomery_form<<<BLOCKS, LIMBS_NUM>>>(a, result);
    assert_checks(expected, result);

    // Montgomery Consistency Check Test
    initialize_montgomery_consistency_check<<<BLOCKS, THREADS>>>(a, b);
    hipDeviceSynchronize();
    montgomery_consistency_check<<<BLOCKS, LIMBS_NUM>>>(a, b, expected, result);
    assert_checks(expected, result);

    // Add Multiplication Consistency Test
    initialize_add_mul_consistency<<<BLOCKS, THREADS>>>(a, b);
    hipDeviceSynchronize();
    add_mul_consistency<<<BLOCKS, LIMBS_NUM>>>(a, b, expected, result);
    assert_checks(expected, result);

    // Subtract Multiplication Consistency test
    initialize_sub_mul_consistency<<<BLOCKS, THREADS>>>(a, b);
    hipDeviceSynchronize();
    sub_mul_consistency<<<BLOCKS, LIMBS_NUM>>>(a, b, expected, result);
    assert_checks(expected, result);

    // Cube Root Test
    initialize_cube<<<BLOCKS, THREADS>>>(a);
    hipDeviceSynchronize();
    cube<<<BLOCKS, LIMBS_NUM>>>(a, expected, result);
    assert_checks(expected, result);
}

/* -------------------------- Main Entry Function ---------------------------------------------- */

int main(int, char**) {
    // Start timer
    auto start = high_resolution_clock::now();

    // Define pointers to uint64_t type
    var *a, *b, *expected, *result;

    // Allocate unified memory accessible by host and device
    hipMallocManaged(&a, LIMBS_NUM * sizeof(var));
    hipMallocManaged(&b, LIMBS_NUM * sizeof(var));
    hipMallocManaged(&expected, LIMBS * sizeof(var));
    hipMallocManaged(&result, LIMBS * sizeof(var));

    // Execute kernel functions
    execute_kernels(a, b, expected, result);

    // Successfull execution of unit tests
    cout << "******* All 'Fq' unit tests passed! **********" << endl;

    // End timer
    auto stop = high_resolution_clock::now();

    // Calculate duraion of execution time 
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time taken by function: " << duration.count() << " microseconds\n" << endl; 

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(result);

    cout << "Completed sucessfully!" << endl;

    return 0;
}