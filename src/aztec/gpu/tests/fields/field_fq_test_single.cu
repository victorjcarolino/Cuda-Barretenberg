#include "hip/hip_runtime.h"
#include "field_single.cu"
#include <assert.h> 

using namespace std;
using namespace std::chrono;
using namespace gpu_barretenberg_single;

static constexpr int LIMBS_NUM = 4; // converted from size_t to eliminate warnings
static constexpr size_t BLOCKS = 1;
static constexpr size_t THREADS = 1;

/* -------------------------- Montgomery Multiplication Test ---------------------------------------------- */

__global__ void initialize_mont_mult(uint254 &a, uint254 &b, uint254 &expected) {
    a = { 0x2523b6fa3956f038, 0x158aa08ecdd9ec1d, 0xf48216a4c74738d4, 0x2514cc93d6f0a1bf };
    b = { 0xb68aee5e4c8fc17c, 0xc5193de7f401d5e8, 0xb8777d4dde671db3, 0xe513e75c087b0bb };
    expected = { 0x7ed4174114b521c4, 0x58f5bd1d4279fdc2, 0x6a73ac09ee843d41, 0x687a76ae9b3425c };
}

__global__ void mont_mult(uint254 a, uint254 b, uint254 &result) {
    fq_single::mul(a, b, result);
}

/* -------------------------- Montgomery Multiplication Test -- Short Integers ---------------------------------------------- */

__global__ void initialize_mont_mult_short(uint254 &a, uint254 &b, uint254 &expected) {
    a = { 0xa, 0, 0, 0 };
    b = { 0xb, 0, 0, 0 };
    expected = { 0x65991a6dc2f3a183, 0xe3ba1f83394a2d08, 0x8401df65a169db3f, 0x1727099643607bba };
}

// duplicate code
// __global__ void mont_mult_short_single(uint254 &a, uint254 &b, uint254 &result) {
//     fq_single::mul(a, b, &result);
// }

/* -------------------------- Multiply - Square Consistency ---------------------------------------------- */

__global__ void initialize_mul_square_consistency(uint254 &a, uint254 &b) {
    a = { 0x329596aa978981e8, 0x8542e6e254c2a5d0, 0xc5b687d82eadb178, 0x2d242aaf48f56b8a };
    b = { 0x7d2e20e82f73d3e8, 0x8e50616a7a9d419d, 0xcdc833531508914b, 0xd510253a2ce62c };
}

__global__ void mul_square_consistency(uint254 a, uint254 b, uint254 &expected, uint254 &result) {
    uint254 t1;
    uint254 t2;

    fq_single::sub(a, b, result);
    t1 = result;
    fq_single::add(a, b, result);
    t2 = result;
    fq_single::mul(t1, t2, expected);

    fq_single::square(a, result);
    t1 = result;
    fq_single::square(b, result);
    t2 = result;
    fq_single::sub(t1, t2, result);
}

/* -------------------------- Multiply - Square Against Constants ---------------------------------------------- */

__global__ void initialize_sqr_check_against_constants(uint254 &a, uint254 &expected) {
    a = { 0x329596aa978981e8, 0x8542e6e254c2a5d0, 0xc5b687d82eadb178, 0x2d242aaf48f56b8a };
    expected = { 0xbf4fb34e120b8b12, 0xf64d70efbf848328, 0xefbb6a533f2e7d89, 0x1de50f941425e4aa };
}

__global__ void sqr_check_against_constants(uint254 a, uint254 &result) {
    fq_single::square(a, result);
}

/* -------------------------- Add - Check Against Constants ---------------------------------------------- */

__global__ void initialize_add_check_against_constants(uint254 &a, uint254 &b, uint254 &expected) {
    a = { 0x7d2e20e82f73d3e8, 0x8e50616a7a9d419d, 0xcdc833531508914b, 0xd510253a2ce62c };
    b = { 0x2829438b071fd14e, 0xb03ef3f9ff9274e, 0x605b671f6dc7b209, 0x8701f9d971fbc9 };
    expected = { 0xa55764733693a536, 0x995450aa1a9668eb, 0x2e239a7282d04354, 0x15c121f139ee1f6 };
}

__global__ void add_check_against_constants(uint254 a, uint254 b, uint254 &result) {
    fq_single::add(a, b, result);
}

/* -------------------------- Subtract - Check Against Constants ---------------------------------------------- */

__global__ void initialize_sub_check_against_constants(uint254 &a, uint254 &b, uint254 &expected) {
    a = { 0xd68d01812313fb7c, 0x2965d7ae7c6070a5, 0x08ef9af6d6ba9a48, 0x0cb8fe2108914f53 };
    b = { 0x2cd2a2a37e9bf14a, 0xebc86ef589c530f6, 0x75124885b362b8fe, 0x1394324205c7a41d };
    expected = { 0xe5daeaf47cf50779, 0xd51ed34a5b0d0a3c, 0x4c2d9827a4d939a6, 0x29891a51e3fb4b5f };
}

__global__ void sub_check_against_constants(uint254 a, uint254 b, uint254 &result) {
    fq_single::sub(a, b, result);
}

/* -------------------------- Convert To Montgomery Form ---------------------------------------------- */

__global__ void initialize_to_montgomery_form(uint254 &a, uint254 &expected) {
    a = { 0x01, 0x00, 0x00, 0x00 };
    expected = { 0xd35d438dc58f0d9d, 0xa78eb28f5c70b3d, 0x666ea36f7879462c, 0xe0a77c19a07df2f };

}

__global__ void to_montgomery_form(uint254 &a, uint254 &result) {
    fq_single::to_monty(a, result);
}

/* -------------------------- Convert From Montgomery Form ---------------------------------------------- */

__global__ void initialize_from_montgomery_form(uint254 &a, uint254 &expected) {
    a = { 0x01, 0x00, 0x00, 0x00 };
    expected = { 0x01, 0x00, 0x00, 0x00 };
}

__global__ void from_montgomery_form(uint254 &a, uint254 &result) {
    fq_single::to_monty(a, result);
    fq_single::from_monty(result, result);
}

/* -------------------------- Montgomery Consistency Check ---------------------------------------------- */

__global__ void initialize_montgomery_consistency_check(uint254 &a, uint254 &b) {
    a = { 0x2523b6fa3956f038, 0x158aa08ecdd9ec1d, 0xf48216a4c74738d4, 0x2514cc93d6f0a1bf };
    b = { 0xb68aee5e4c8fc17c, 0xc5193de7f401d5e8, 0xb8777d4dde671db3, 0xe513e75c087b0bb };
}

__global__ void montgomery_consistency_check(uint254 &a, uint254 &b, uint254 &expected, uint254 &result) {
    uint254 aR;
    uint254 bR;
    uint254 aRR;
    uint254 bRR;
    uint254 bRRR;
    uint254 result_a;
    uint254 result_b;
    uint254 result_c;
    uint254 result_d;

    fq_single::to_monty(a, aR);
    fq_single::to_monty(aR, aRR);
    fq_single::to_monty(b, bR);
    fq_single::to_monty(bR, bRR);
    fq_single::to_monty(bRR, bRRR);

    fq_single::mul(aRR, bRR, result_a); // abRRR
    fq_single::mul(aR, bRRR, result_b); // abRRR
    // fq_single::mul(aR, bR, result_c);   // abR
    fq_single::mul(a, b, result_d);               // abR^-1

    fq_single::from_monty(result_a, result_a);    // abRR
    fq_single::from_monty(result_a, result_a);    // abR
    fq_single::from_monty(result_a, result);    // ab
    // fq_single::from_monty(result_c, result_c);    // ab
    fq_single::to_monty(result_d, expected);      // ab
}

/* -------------------------- Add Multiplication Consistency ---------------------------------------------- */

__global__ void initialize_add_mul_consistency(uint254 &a, uint254 &b) {
    a = { 0x2523b6fa3956f038, 0x158aa08ecdd9ec1d, 0xf48216a4c74738d4, 0x2514cc93d6f0a1bf };
    b = { 0x09, 0, 0, 0 };
}

__global__ void add_mul_consistency(uint254 &a, uint254 &b, uint254 &expected, uint254 &res) {
    uint254 multiplicand;

    fq_single::to_monty(b, multiplicand);
    fq_single::add(a, a, res);                       // 2
    fq_single::add(res, res, res);             // 4
    fq_single::add(res, res, res);             // 8
    fq_single::add(res, a, res);                  // 9

    fq_single::mul(a, multiplicand, expected);                // 9
}

/* -------------------------- Subtract Multiplication Consistency ---------------------------------------------- */

__global__ void initialize_sub_mul_consistency(uint254 &a, uint254 &b) {
    a = { 0x2523b6fa3956f038, 0x158aa08ecdd9ec1d, 0xf48216a4c74738d4, 0x2514cc93d6f0a1bf };
    b = { 0x05, 0, 0, 0 };
}

__global__ void sub_mul_consistency(uint254 &a, uint254 &b, uint254 &expected, uint254 &res) {
    uint254 multiplicand;

    fq_single::to_monty(b, multiplicand);
    fq_single::add(a, a, res);                       // 2
    fq_single::add(res, res, res);             // 4
    fq_single::add(res, res, res);             // 8
    fq_single::sub(res, a, res);                  // 7
    fq_single::sub(res, a, res);                  // 6
    fq_single::sub(res, a, res);                  // 5

    fq_single::mul(a, multiplicand, expected);                // 5
}

/* -------------------------- Cube Root ---------------------------------------------- */

__global__ void initialize_cube(uint254 &a) {
    a = { 0x2523b6fa3956f038, 0x158aa08ecdd9ec1d, 0xf48216a4c74738d4, 0x2514cc93d6f0a1bf };
}

__global__ void cube(uint254 &a, uint254 &expected, uint254 &result) {
    uint254 x_cubed;
    uint254 beta_x;
    uint254 beta_x_cubed;

    fq_single::mul(a, a, result);
    x_cubed = result;
    fq_single::mul(x_cubed, a, result);
    x_cubed = result;
    
    fq_single::mul(a, gpu_barretenberg_single::CUBE_ROOT_BASE, expected);
    beta_x = expected;
    fq_single::mul(beta_x, beta_x, expected);
    beta_x_cubed = expected;
    fq_single::mul(beta_x_cubed, beta_x, expected);
    beta_x_cubed = expected;
}

/* -------------------------- Executing Initialization and Workload Kernels ---------------------------------------------- */

void assert_checks(uint254 *expected, uint254 *result) {
    // Explicit synchronization barrier
    hipDeviceSynchronize();

    // Print statements
    for(int i=0; i<LIMBS_NUM; i++) {
        // printf("expected->limbs[%d] is: %zu\n", i, expected->limbs[i]);
        printf("expected->limbs[%d] is: %lx\n", i, expected->limbs[i]);
    }
    for(int i=0; i<LIMBS_NUM; i++) {
        // printf("result->limbs[%d] is: %zu\n", i, result->limbs[i]);
        printf("result->limbs[%d] is: %lx\n", i, result->limbs[i]);
    }
    // Assert clause    
    for(int i=0; i<LIMBS_NUM; i++) {
        if (expected->limbs[i] != result->limbs[i]) {
            printf("***** BAD!!!\n");
        }
        assert(expected->limbs[i] == result->limbs[i]);
    }
}

void execute_kernels(uint254 *a, uint254 *b, uint254 *expected, uint254 *result) {    
    // Montgomery Multiplication Test 
    printf("\n*** mont_mult ***\n");
    initialize_mont_mult<<<BLOCKS, THREADS>>>(*a, *b, *expected);
    // printf("\n??? zz %lu %lu %lu %lu\n", a->limbs[0], a->limbs[1], a->limbs[2], a->limbs[3]);
    hipDeviceSynchronize();
    mont_mult<<<BLOCKS, THREADS>>>(*a, *b, *result);
    assert_checks(expected, result);

    // Montgomery Multiplication Test -- Short Integers 
    printf("\n*** mont_mult short ***\n");
    initialize_mont_mult_short<<<BLOCKS, THREADS>>>(*a, *b, *expected);
    hipDeviceSynchronize();
    mont_mult<<<BLOCKS, THREADS>>>(*a, *b, *result);
    assert_checks(expected, result);

    // Multiply Test - Square Consistency 
    printf("\n*** mont_mult sq consistency ***\n");
    initialize_mul_square_consistency<<<BLOCKS, THREADS>>>(*a, *b);
    hipDeviceSynchronize();
    mul_square_consistency<<<BLOCKS, THREADS>>>(*a, *b, *expected, *result);
    assert_checks(expected, result);

    // Multiply Test - Square Against Constants 
    printf("\n*** mont_mult sq against constants ***\n");
    initialize_sqr_check_against_constants<<<BLOCKS, THREADS>>>(*a, *expected);
    hipDeviceSynchronize();
    sqr_check_against_constants<<<BLOCKS, THREADS>>>(*a, *result);
    assert_checks(expected, result);

    // Add Test - Check Against Constants
    printf("\n*** add check against constants ***\n");
    initialize_add_check_against_constants<<<BLOCKS, THREADS>>>(*a, *b, *expected);
    hipDeviceSynchronize();
    add_check_against_constants<<<BLOCKS, THREADS>>>(*a, *b, *result);
    assert_checks(expected, result);

    // Subtract Test - Check Against Constant
    printf("\n*** sub check against constants ***\n");
    initialize_sub_check_against_constants<<<BLOCKS, THREADS>>>(*a, *b, *expected);
    hipDeviceSynchronize();
    sub_check_against_constants<<<BLOCKS, THREADS>>>(*a, *b, *result);
    assert_checks(expected, result);

    // Convert To Montgomery Form Test
    printf("\n*** to monty ***\n");
    initialize_to_montgomery_form<<<BLOCKS, THREADS>>>(*a, *expected);
    hipDeviceSynchronize();
    to_montgomery_form<<<BLOCKS, THREADS>>>(*a, *result);
    assert_checks(expected, result);

    // Convert From Montgomery Form Test
    printf("\n*** from monty ***\n");
    initialize_from_montgomery_form<<<BLOCKS, THREADS>>>(*a, *expected);
    hipDeviceSynchronize();
    from_montgomery_form<<<BLOCKS, THREADS>>>(*a, *result);
    assert_checks(expected, result);

    // Montgomery Consistency Check Test
    printf("\n*** monty consistency ***\n");
    initialize_montgomery_consistency_check<<<BLOCKS, THREADS>>>(*a, *b);
    hipDeviceSynchronize();
    montgomery_consistency_check<<<BLOCKS, THREADS>>>(*a, *b, *expected, *result);
    assert_checks(expected, result);

    // Add Multiplication Consistency Test
    printf("\n*** add mul consistency ***\n");
    initialize_add_mul_consistency<<<BLOCKS, THREADS>>>(*a, *b);
    hipDeviceSynchronize();
    add_mul_consistency<<<BLOCKS, THREADS>>>(*a, *b, *expected, *result);
    assert_checks(expected, result);

    // Subtract Multiplication Consistency test
    printf("\n*** sub mul consistency ***\n");
    initialize_sub_mul_consistency<<<BLOCKS, THREADS>>>(*a, *b);
    hipDeviceSynchronize();
    sub_mul_consistency<<<BLOCKS, THREADS>>>(*a, *b, *expected, *result);
    assert_checks(expected, result);

    // Cube Root Test
    printf("\n*** cube root ***\n");
    initialize_cube<<<BLOCKS, THREADS>>>(*a);
    hipDeviceSynchronize();
    cube<<<BLOCKS, THREADS>>>(*a, *expected, *result);
    assert_checks(expected, result);
}

/* -------------------------- Main Entry Function ---------------------------------------------- */

int main(int, char**) {
    // Start timer
    auto start = high_resolution_clock::now();

    // Define uint256 types
    uint254 *a, *b, *expected, *result;

    // Allocate unified memory accessible by host and device
    hipMallocManaged(&a, LIMBS_NUM * sizeof(var));
    hipMallocManaged(&b, LIMBS_NUM * sizeof(var));
    hipMallocManaged(&expected, LIMBS * sizeof(var));
    hipMallocManaged(&result, LIMBS * sizeof(var));

    // Execute kernel functions
    execute_kernels(a, b, expected, result);

    // Successfull execution of unit tests
    cout << "******* All 'Fq' unit tests passed! **********" << endl;

    // End timer
    auto stop = high_resolution_clock::now();

    // Calculate duraion of execution time 
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time taken by function: " << duration.count() << " microseconds\n" << endl; 

    // Free unified memory
    hipFree(&a);
    hipFree(&b);
    hipFree(&result);

    cout << "Completed sucessfully!" << endl;

    return 0;
}